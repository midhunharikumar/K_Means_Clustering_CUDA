#include "hip/hip_runtime.h"
/*

K- Means Clustering in GPU
@Author Midhun Harikumar
@Date   10/23/2015


Function performs K- Means clustering on different Data point and Cluster inputs
for the CUDA capable NVIDIA GPU



*/





#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include "KMeans.h"


__constant__ Vector2 clusterD[3];








__global__ void findDistances(Datapoint *dataD, int n, int k){

	double dist[3] = { 5000, 500, 5000 };
	double distmin = 5000;
	int minval = 0;

	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int tid = blockId * (blockDim.x * blockDim.y)
		+ (threadIdx.y * blockDim.x) + threadIdx.x;
	//int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	/*int tid = blockIdx.x * blockDim.x * blockDim.y
		+ threadIdx.y * blockDim.x + threadIdx.x;*/

	if (tid < n){

		dist[0] = clusterD[0].distSq(dataD[tid].p);
		dist[1] = clusterD[1].distSq(dataD[tid].p);
		dist[2] = clusterD[2].distSq(dataD[tid].p);

		for (int i = 0; i < 3; i++){

			if (distmin>dist[i]){
				minval = i;

				distmin = dist[i];
			}


		}

		if (minval != dataD[tid].cluster){
			dataD[tid].cluster = minval;

			dataD[tid].altered = true;
		}
		else{
			dataD[tid].altered = false;
		}




	}//if loopClose




}








bool KMeansGPU(Datapoint* data, long n, Vector2* clusters, int k){


	hipError_t status;

	int data_bytes = sizeof(Datapoint)*n;
	int cluster_bytes = sizeof(Vector2)*k;
	int altered_data_cnt = 1555;
	int clusterCount = 0;
	Datapoint* dataD;




	// Allocate memory on device
	hipMalloc((void**)&dataD, data_bytes);
	hipMemcpyToSymbol(HIP_SYMBOL(clusterD), clusters, cluster_bytes);
	hipMemcpyFromSymbol(clusters, HIP_SYMBOL(clusterD), cluster_bytes);

	// Copy data to allocated memory	
	hipMemcpy(dataD, data, data_bytes, hipMemcpyHostToDevice);

	// Set Grid and block Dimensions
	dim3 dimblock(16, 16, 1);
	dim3 dimgrid;
	dimgrid.x = ceil(sqrt((float)n / 256));
	dimgrid.y = ceil(sqrt((float)n / 256));

	// Check state	
	while (altered_data_cnt != 0){
		altered_data_cnt = 0;
		// Find distances 
		findDistances << <dimgrid, dimblock >> >(dataD, n, k);
		hipDeviceSynchronize();


		// Copy Data back
		hipMemcpy(data, dataD, data_bytes, hipMemcpyDeviceToHost);



		for (int i = 0; i < k; i++){

			for (int j = 0; j < n; j++){

				if (data[j].cluster == i){

					clusters[i].x += data[j].p.x;
					clusters[i].y += data[j].p.y;
					clusterCount++;
				}

			}
			if (clusterCount != 0){
				clusters[i].x /= clusterCount;
				clusters[i].y /= clusterCount;
			}
			clusterCount = 0;
		}

		for (int i = 0; i < n; i++){

			if (data[i].altered){
				altered_data_cnt++;
				data[i].altered = false;
			}

		}

		hipMemcpy(dataD, data, data_bytes, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(clusterD), clusters, cluster_bytes);

	}









	status = hipGetLastError();
	if (status != hipSuccess) {
		std::cout << "Kernel failed: " << hipGetErrorString(status) <<
			std::endl;
		hipFree(dataD);


		return false;
	}




	hipFree(dataD);


	return true;
}